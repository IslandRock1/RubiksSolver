
#include <hip/hip_runtime.h>
#include <iostream>
#include <array>

// Device code
__global__ void VecAdd(float* A, float* B, float* C, int N) {
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if (i < N)
		C[i] = A[i] + B[i];
}

// Host code
int main() {
	int N = 2048;
	size_t size = N * sizeof(float);

	float* h_B = (float*)malloc(size);
	float* h_C = (float*)malloc(size);
	float* h_A = (float*)malloc(size);

	// Initialize input vectors
	h_A[0] = 69;
	h_B[0] = 2;

	std::cout << h_C[0] << "\n";

	// Allocate vectors in device memory
	float* d_A;
	hipMalloc(&d_A, size);
	float* d_B;
	hipMalloc(&d_B, size);
	float* d_C;
	hipMalloc(&d_C, size);

	// Copy vectors from host memory to device memory
	hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
	hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

	// Invoke kernel
	int threadsPerBlock = 256;
	int blocksPerGrid =
			(N + threadsPerBlock - 1) / threadsPerBlock;
	VecAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);

	// Copy result from device memory to host memory
	// h_C contains the result in host memory
	hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

	std::cout << h_C[0] << "\n";

	// Free device memory
	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);


}
